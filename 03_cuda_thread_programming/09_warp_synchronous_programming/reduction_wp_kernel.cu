#include "hip/hip_runtime.h"
#include <stdio.h>
#include "reduction.h"

#define FULL_MASK 0xffffffff

/*
    Parallel sum reduction using shared memory
    - takes log(n) steps for n input elements
    - uses n threads
    - only works for power-of-2 arrays
*/

/**
    Two warp level primitives are used here for this example
    https://devblogs.nvidia.com/faster-parallel-reductions-kepler/
    https://devblogs.nvidia.com/using-cuda-warp-level-primitives/
 */

__inline__ __device__ float warp_reduce_sum(float val)
{
    for (int offset = warpSize / 2; offset > 0; offset >>= 1)
        val += __shfl_down_sync(FULL_MASK, val, offset);
    return val;
}

__inline__ __device__ float block_reduce_sum(float val)
{
    static __shared__ float shared[32]; // Shared mem for 32 partial sums
    int lane = threadIdx.x % warpSize;
    int wid = threadIdx.x / warpSize;

    val = warp_reduce_sum(val); // Each warp performs partial reduction

    if (lane == 0)
        shared[wid] = val; // Write reduced value to shared memory

    __syncthreads(); // Wait for all partial reductions

    //read from shared memory only if that warp existed
    if (wid == 0) {
        val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0;
        val = warp_reduce_sum(val); //Final reduce within first warp
    }

    return val;
}

// cuda thread synchronization
__global__ void
reduction_kernel(float *g_out, float *g_in, unsigned int size)
{
    unsigned int idx_x = blockIdx.x * (2 * blockDim.x) + threadIdx.x;

    float sum = 0.f;
    // reduce one more data
    sum += (idx_x < size) ? g_in[idx_x] : 0.f;
    sum += ((idx_x + blockDim.x) < size) ? g_in[idx_x + blockDim.x] : 0.f;

    sum = block_reduce_sum(sum);

    if (threadIdx.x == 0)
        g_out[blockIdx.x] = sum;
}

int reduction(float *g_outPtr, float *g_inPtr, int size, int n_threads)
{
    int block_size = 2 * n_threads;
    int n_blocks = (size + block_size - 1) / block_size;
    reduction_kernel<<<n_blocks, n_threads>>>(g_outPtr, g_inPtr, size);
    return n_blocks;
}
