#include "hip/hip_runtime.h"
#include <stdio.h>
#include "reduction.h"

#define FULL_MASK 0xffffffff

/*
    Parallel sum reduction using shared memory
    - takes log(n) steps for n input elements
    - uses n threads
    - only works for power-of-2 arrays
*/

/**
    Two warp level primitives are used here for this example
    https://devblogs.nvidia.com/faster-parallel-reductions-kepler/
    https://devblogs.nvidia.com/using-cuda-warp-level-primitives/
 */

__inline__ __device__ float warp_reduce_sum(float val)
{
    for (int offset = warpSize / 2; offset > 0; offset >>= 1)
        val += __shfl_down_sync(FULL_MASK, val, offset);
    return val;
}

__inline__ __device__ float block_reduce_sum(float val)
{
    static __shared__ float shared[32]; // Shared mem for 32 partial sums
    int lane = threadIdx.x % warpSize;
    int wid = threadIdx.x / warpSize;

    val = warp_reduce_sum(val); // Each warp performs partial reduction

    if (lane == 0)
        shared[wid] = val; // Write reduced value to shared memory

    __syncthreads(); // Wait for all partial reductions

    //read from shared memory only if that warp existed
    if (wid == 0) {
        val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0;
        val = warp_reduce_sum(val); //Final reduce within first warp
    }

    return val;
}

// cuda thread synchronization
__global__ void
reduction_kernel(float *g_out, float *g_in, unsigned int size)
{
    unsigned int idx_x = blockIdx.x * (2 * blockDim.x) + threadIdx.x;

    // cumulates input with grid-stride loop and save to share memory
    float sum = 0.f;
    for (int i = idx_x; i < size; i += blockDim.x * gridDim.x)
        sum += g_in[i];

    sum = block_reduce_sum(sum);

    if (threadIdx.x == 0)
        g_out[blockIdx.x] = sum;
}

void reduction(float *g_outPtr, float *g_inPtr, int size, int n_threads)
{
    int num_sms;
    int num_blocks_per_sm;
    hipDeviceGetAttribute(&num_sms, hipDeviceAttributeMultiprocessorCount, 0);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks_per_sm, reduction_kernel, n_threads, n_threads*sizeof(float));
    int n_blocks = min(num_blocks_per_sm * num_sms, (size + n_threads - 1) / n_threads);

    reduction_kernel<<<n_blocks, n_threads>>>(g_outPtr, g_inPtr, size);
    reduction_kernel<<< 1, n_threads, n_threads * sizeof(float), 0 >>>(g_outPtr, g_inPtr, n_blocks);
}
