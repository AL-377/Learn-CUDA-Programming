
#include <hip/hip_runtime.h>
#include <cstdio>

using namespace std;

__global__ void vecAdd_kernel(float *c, const float* a, const float* b);

void cuda_async_operation(float *h_c, const float *h_a, const float *h_b,
                          float *d_c, float *d_a, float *d_b,
                          const int size, const int bufsize, hipStream_t stream = 0)
{
    // create cuda event
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // transfer data from host to device
    hipMemcpyAsync(d_a, h_a, bufsize, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_b, h_b, bufsize, hipMemcpyHostToDevice, stream);

    hipEventRecord(start, stream);

    // launch kernel
    dim3 block_size(256);
    dim3 grid_size(size / block_size.x);
    vecAdd_kernel<<< grid_size, block_size, 0, stream >>>(d_c, d_a, d_b);

    hipEventRecord(stop, stream);

    // transfer the result
    hipMemcpyAsync(h_c, d_c, bufsize, hipMemcpyDeviceToHost, stream);

    // synchronization with the event
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("%f ms\n", milliseconds);

    // delete cuda event
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

void init_buffer(float *data, const int size);

int main()
{
    float *h_a, *h_b, *h_c;
    float *d_a, *d_b, *d_c;
    int size = 1 << 16;
    int bufsize = size * sizeof(float);
    hipStream_t stream;

    // allocate host memories
    h_a = new float[size];
    h_b = new float[size];
    h_c = new float[size];

    // initialize host values
    srand(2019);
    init_buffer(h_a, size);
    init_buffer(h_b, size);
    init_buffer(h_c, size);

    // create cuda stream
    hipStreamCreate(&stream);

    // allocate device memories
    hipMalloc((void**)&d_a, bufsize);
    hipMalloc((void**)&d_b, bufsize);
    hipMalloc((void**)&d_c, bufsize);
    
    cuda_async_operation(h_c, h_a, h_b, d_c, d_a, d_b, size, bufsize, stream);

    // print out the result
    int print_idx = 256;
    printf("compared a sample result...\n");
    printf("host: %.6f, device: %.6f\n",  h_a[print_idx] + h_b[print_idx], h_c[print_idx]);

    // terminate cuda stream
    hipStreamDestroy(stream);

    // terminate device memories
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // terminate host memories
    delete [] h_a;
    delete [] h_b;
    delete [] h_c;
    
    return 0;
}

void init_buffer(float *data, const int size)
{
    for (int i = 0; i < size; i++) 
        data[i] = rand() / (float)RAND_MAX;
}

__global__ void
vecAdd_kernel(float *c, const float* a, const float* b)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = 0; i < 500; i++)
       c[idx] = a[idx] + b[idx];
}