#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "helper.cuh"

#define BLOCKDIM 512

template <typename T>
void printMatrix(const T *matrix, const int ldm, const int n);

int main()
{
    hipblasStatus_t stat;
    hipblasHandle_t cublas_handle;

    CBuffer<float> A, B, C;
    int M, N, K;
    float alpha, beta;

    M = 4;
    N = 5;
    K = 6;
    alpha = 1.f;
    beta = 0.f;

    stat = hipblasCreate(&cublas_handle);
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout << "CUBLAS initialization failed" << std::endl;
        return EXIT_FAILURE;
    }

    srand(2019);

    // initialize host buffers
    A.init(K * M, true);
    B.init(N * K, true);
    C.init(N * M, true);

    std::cout << "A:" << std::endl;
    printMatrix(A.h_ptr_, K, M);
    std::cout << "B:" << std::endl;
    printMatrix(B.h_ptr_, N, K);
    std::cout << "C:" << std::endl;
    printMatrix(C.h_ptr_, N, M);

    A.cuda(true);
    B.cuda(true);
    C.cuda(true);

    stat = hipblasGemmEx(cublas_handle,
                        HIPBLAS_OP_N,
                        HIPBLAS_OP_N,
                        M, N, K,
                        &alpha,
                        A.d_ptr_, HIP_R_32F, M,
                        B.d_ptr_, HIP_R_32F, K,
                        &beta,
                        C.d_ptr_, HIP_R_32F, M,
                        HIP_R_32F,
                        HIPBLAS_GEMM_DEFAULT);

    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout << "CUBLAS operation failed [" << stat << "]" << std::endl;
        return EXIT_FAILURE;
    }

    C.copyToHost();

    std::cout << "C out:" << std::endl;
    printMatrix(C.h_ptr_, N, M);

    hipblasDestroy(cublas_handle);

    return 0;
}

template <typename T>
void printMatrix(const T *matrix, const int ldm, const int n)
{
    for (int j = 0; j < n; j++)
    {
        for (int i = 0; i < ldm; i++)
        {
            if (sizeof(T) >= 2)
                std::cout << std::fixed << std::setw(8) << std::setprecision(4) << matrix[IDX2C(i, j, ldm)];
            else
                std::cout << std::fixed << std::setw(4) << static_cast<int16_t>(matrix[IDX2C(i, j, ldm)]);
        }
        std::cout << std::endl;
    }
}