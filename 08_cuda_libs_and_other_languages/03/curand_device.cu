#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>

#include <hipblas.h>

#include <hiprand/hiprand_kernel.h>

#define IDX2C(i, j, ld) (((j) * (ld)) + (i))

template <typename T>
void printMatrix(const T *matrix, const int ldm, const int n);

__global__ void setup_kernel(hiprandState_t *state)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Each thread gets same seed, 
    // a different sequence number, no offset */
    hiprand_init(2019UL, idx, 0, &state[idx]);
}

__global__ void generate_kernel(unsigned int *generated_out, hiprandState_t *state)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    generated_out[idx] = hiprand(&state[idx]);
}

__global__ void generate_uniform_kernel(float *generated_out, hiprandState_t *state)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    generated_out[idx] = hiprand_uniform(&state[idx]);
}

#define BLOCK_DIM 512

/*
 * random integer number generation
 */
void cuRandGenerator(const hiprandState_t *devStates, unsigned int **npHostResult, const size_t length)
{
    unsigned int *npDevResult;

    (*npHostResult) = new unsigned int[length];
    hipMalloc((void **)&npDevResult, length * sizeof(unsigned int));

    // random number generation
    generate_kernel<<<(length + BLOCK_DIM - 1) / BLOCK_DIM, BLOCK_DIM>>>(npDevResult, const_cast<hiprandState_t *>(devStates));
    hipMemcpy(*npHostResult, npDevResult, length * sizeof(unsigned int), hipMemcpyDeviceToHost);

    // terminates memory
    hipFree(npDevResult);
}

/*
 * Random uniform distributed floating number generation
 */
void cuRandUniformGenerator(const hiprandState_t *devStates, float **fpHostResult, const size_t length)
{
    float *fpDevResult;

    (*fpHostResult) = new float[length];
    hipMalloc((void **)&fpDevResult, length * sizeof(float));

    // random number generation
    generate_uniform_kernel<<<(length + BLOCK_DIM - 1) / BLOCK_DIM, BLOCK_DIM>>>(fpDevResult, const_cast<hiprandState_t *>(devStates));
    hipMemcpy(*fpHostResult, fpDevResult, length * sizeof(float), hipMemcpyDeviceToHost);

    // terminates device memory
    hipFree(fpDevResult);
}

int main(int argc, char *argv[])
{
    hiprandState_t *devStates;
    unsigned int *npHostResult;
    float *fpHostResult;
    int opt = 0;

    int M = 4, N = 5;
    size_t length = M * N;

    // Select random number generation option
    if (argc == 2) 
        opt = atoi(argv[1]);

    // allcate space for prng states on device
    hipMalloc((void **)&devStates, length * sizeof(hiprandState));

    /* Initialize the states */
    setup_kernel<<<(length + BLOCK_DIM - 1) / BLOCK_DIM, BLOCK_DIM>>>(devStates);

    if (opt == 0)
    {
        std::cout << "Generated random numbers" << std::endl;
        cuRandGenerator(devStates, &npHostResult, length);
        printMatrix(npHostResult, M, N);
        delete [] npHostResult;
    }
    else
    {
        std::cout << "Generated uniform random numbers" << std::endl;
        cuRandUniformGenerator(devStates, &fpHostResult, length);
        printMatrix(fpHostResult, M, N);
        delete [] fpHostResult;
    }

    // terminates curand device states
    hipFree(devStates);

    return 0;
}

template <typename T>
void printMatrix(const T *matrix, const int ldm, const int n)
{
    for (int j = 0; j < n; j++)
    {
        for (int i = 0; i < ldm; i++)
            std::cout << std::fixed << std::setw(12) << std::setprecision(4) << matrix[IDX2C(i, j, ldm)];
        std::cout << std::endl;
    }
}