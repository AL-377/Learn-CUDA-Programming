#include "hip/hip_runtime.h"
#include <stdio.h>
#include <helper_timer.h>
#include <hip/hip_runtime_api.h>

#define BLOCK_DIM 16

////////////////////////////////////////////////////////////////////////////////
//! Compute reference data set matrix multiply on GPU
//! C = alpha * A * B + beta * C
//! @param A          matrix A as provided to device (M x K)
//! @param B          matrix B as provided to device (K x N)
//! @param C          matrix C as provided to device (M x N)
//! @param N          height of matrix A and matrix C
//! @param M          width of matrix B and matrix C
//! @param K          width of matrix A and height of matrix C
//! @param alpha      scala value for matrix multiplication
//! @param beta       scala value for matrix summation with C
////////////////////////////////////////////////////////////////////////////////

__global__ void sgemm_kernel(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    float element_c = 0.f;
    for (int e = 0; e < K; e++)
        element_c += A[row * K + e] * B[e * K + col];

    C[row * N + col] = alpha * element_c + beta * C[row * N + col];
}

void sgemm_gold(const float *A, const float *B, float *C, int M, int N, int K, float alpha, float beta)
{
    float element_c = 0.f;
    for (int row = 0; row < M; row++)
    {
        for (int col = 0; col < N; col++)
        {
            for (int e = 0; e < K; e++)
	    {
                element_c += A[row * K + e] * B[e * N + col];
	    }
            C[row * N + col] = alpha * element_c + beta * C[row * N + col];
        }
    }
}

void random_init(float *data, int length)
{
    for (int i = 0; i < length; i++)
    {
        data[i] = (rand() & 0xFF) / (float)RAND_MAX;
    }
}

bool value_test(float *a, float *b, int length)
{
    float epsilon = 0.000001;
    for (int i = 0; i < length; i++)
        if (abs(a[i] - b[i]) >= epsilon)
            return false;
    return true;
}

int main()
{
    float *A, *B, *C_host, *C_gpu;
    float *d_A, *d_B, *d_C;
    int N, M, K;
    float alpha = 2.f;
    float beta = 1.f;
    int n_iter = 5;
    N = M = K = 2048;

    // initialize timer
    StopWatchInterface *timer;
    sdkCreateTimer(&timer);

    // allocation of linear memory space
    A = (float *)malloc(N * K * sizeof(float));
    B = (float *)malloc(K * M * sizeof(float));
    C_host = (float *)malloc(M * N * sizeof(float));
    C_gpu = (float *)malloc(M * N * sizeof(float));

    // allocation of gpu linear memory space
    hipMalloc((void **)&d_A, N * K * sizeof(float));
    hipMalloc((void **)&d_B, K * M * sizeof(float));
    hipMalloc((void **)&d_C, M * N * sizeof(float));

    // initialize randomized values for memory space
    random_init(A, N * K);
    random_init(B, K * M);

    // profiler will focus from this point
    sdkStartTimer(&timer);

    // copy initial value for gpu memory
    hipMemcpy(d_A, A, N * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, A, K * M * sizeof(float), hipMemcpyHostToDevice);

    // do operation
    dim3 blockDim(BLOCK_DIM, BLOCK_DIM);
    dim3 gridDim((N + BLOCK_DIM - 1) / BLOCK_DIM, (M + BLOCK_DIM - 1) / BLOCK_DIM);
    hipProfilerStart();
    for (int i = 0; i < n_iter; i++) {
        sgemm_kernel<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, N, K, alpha, beta);
    }
    // measuring the performance
    hipDeviceSynchronize();
    sdkStopTimer(&timer); // this profiler should be behined of device synchronization

    // copy data from the gpu
    hipMemcpy(C_gpu, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);

    // profiler will stop its focus
    hipProfilerStop();

    // compare the result
    sgemm_gold(A, B, C_host, M, N, K, alpha, beta);
    if (value_test(C_host, C_gpu, M * N))
        printf("SUCCESS!!\n");
    else
        printf("Error\n");

    // terminates allocated gpu memory space
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // terminates allocated memory space
    free(A);
    free(B);
    free(C_host);
    free(C_gpu);

    return 0;
}